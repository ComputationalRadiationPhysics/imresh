/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2016 Philipp Trommler
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <functional>               // std::function
#ifdef IMRESH_DEBUG
#   include <iostream>              // std::cout, std::endl
#endif
#include <list>                     // std::list
#include <mutex>                    // std::mutex
#include <thread>                   // std::thread
#include <utility>                  // std::pair
#include <cassert>

#include "algorithms/cuda/cudaShrinkWrap.h"
#include "libs/cudacommon.h"        // CUDA_ERROR

namespace imresh
{
namespace io
{

    /**
     * Struct containing a CUDA stream with it's associated device.
     */
    struct stream
    {
        int device;
        hipStream_t str;
    };

    /**
     * Mutex to coordinate device usage.
     */
    std::mutex mtx;
    /**
     * List where all streams are stored as imresh::io::stream structs.
     */
    std::list<stream> streamList;
    /**
     * List to store all created threads.
     */
    std::list<std::thread> threadPool;
    /**
     * Maximum size of the thread pool.
     *
     * This is determined while imresh::io::fillStreamList() as the number of
     * available streams.
     */
    unsigned int threadPoolMaxSize = 0;

    /**
     * Function to add a image processing task to the queue.
     *
     * This is called from taskQueue::addTask() as a thread to prevent blocking
     * and to ensure that all streams are filled with work. It selects the least
     * recently used stream from the streamList and fills it with new work (FIFO).
     *
     * A mutex ensures the correct work balancing over the CUDA streams.
     * However, this mutex doesn't include the call to the write out function.
     * If you need your write out function to be thread safe, you'll have to
     * use your own lock mechanisms inside of this function.
     *
     * @see addTask
     */
    void addTaskAsync(
        float* _h_mem,
        std::pair<unsigned int,unsigned int> _size,
        std::function<void(float*,std::pair<unsigned int,unsigned int>,
            std::string)> _writeOutFunc,
        std::string _filename,
        unsigned int _numberOfCycles,
        unsigned int _numberOfHIOCycles,
        float _targetError,
        float _HIOBeta,
        float _intensityCutOffAutoCorel,
        float _intensityCutOff,
        float _sigma0,
        float _sigmaChange
    )
    {
        // Lock the mutex so no other thread intermediatly changes the device
        // selection
        mtx.lock( );
        // Get the next device and stream to use
        auto strm = streamList.front( );
        streamList.pop_front( );
        streamList.push_back( strm );
        auto device = strm.device;
        auto str = strm.str;

        // Select device and copy memory
        CUDA_ERROR( hipSetDevice( device ) );

#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::addTaskAsync(): Mutex locked, device and stream selected. Calling shrink-wrap."
                << std::endl;
#       endif

        // Call shrinkWrap in the selected stream on the selected device.
        imresh::algorithms::cuda::cudaShrinkWrap( _h_mem,
                                              _size.first,
                                              _size.second,
                                              str,
                                              _numberOfCycles,
                                              _targetError,
                                              _HIOBeta,
                                              _intensityCutOffAutoCorel,
                                              _intensityCutOff,
                                              _sigma0,
                                              _sigmaChange,
                                              _numberOfHIOCycles );

        mtx.unlock( );

#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::addTaskAsync(): CUDA work finished, mutex unlocked. Calling write out function."
                << std::endl;
#       endif

        _writeOutFunc( _h_mem, _size, _filename );
    }

    void addTask(
        float* _h_mem,
        std::pair<unsigned int,unsigned int> _size,
        std::function<void(float*,std::pair<unsigned int,unsigned int>,
            std::string)> _writeOutFunc,
        std::string _filename,
        unsigned int _numberOfCycles = 20,
        unsigned int _numberOfHIOCycles = 20,
        float _targetError = 0.00001f,
        float _HIOBeta = 0.9f,
        float _intensityCutOffAutoCorel = 0.04f,
        float _intensityCutOff = 0.2f,
        float _sigma0 = 3.0f,
        float _sigmaChange = 0.01f
    )
    {
        assert( threadPoolMaxSize > 0 and "Did you make a call to taskQueueInit?" );

        while( threadPool.size( ) >= threadPoolMaxSize )
        {
#           ifdef IMRESH_DEBUG
                std::cout << "imresh::io::addTask(): Too many active threads. Waiting for one of them to finish."
                    << std::endl;
#           endif
            if ( threadPool.front().joinable() )
                threadPool.front( ).join( );
            else
            {
#               ifdef IMRESH_DEBUG
                    std::cout << "[Warning] " << __FILE__ << " line " << __LINE__
                              << ": a thread from the thread pool is not joinable!\n";
#               endif
            }
            threadPool.pop_front( );
        }

#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::addTask(): Creating working thread."
                << std::endl;
#       endif

        threadPool.push_back( std::thread( addTaskAsync, _h_mem,
                                                         _size,
                                                         _writeOutFunc,
                                                         _filename,
                                                         _numberOfCycles,
                                                         _numberOfHIOCycles,
                                                         _targetError,
                                                         _HIOBeta,
                                                         _intensityCutOffAutoCorel,
                                                         _intensityCutOff,
                                                         _sigma0,
                                                         _sigmaChange ) );
    }

    /**
     * This function adds all streams to the stream list.
     *
     * To achieve that it iterates over all available devices and creates one
     * stream for each multiprocessor on each device. All these streams are
     * stored in the streamList as imresh::io::stream objects. If no streams are
     * found, the program aborts.
     */
    unsigned fillStreamList( )
    {
#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::fillStreamList(): Starting stream creation."
                << std::endl;
#       endif
        int deviceCount = 0;
        CUDA_ERROR( hipGetDeviceCount( &deviceCount ) );

        if( deviceCount <= 0 )
        {
#           ifdef IMRESH_DEBUG
                std::cout << "imresh::io::fillStreamList(): No devices found. Aborting."
                    << std::endl;
#           endif
            exit( EXIT_FAILURE );
        }

        for( int i = 0; i < deviceCount; i++ )
        {
            hipDeviceProp_t prop;
            CUDA_ERROR( hipGetDeviceProperties( &prop, i ) );

            assert( prop.multiProcessorCount >= 0 );
#           ifdef IMRESH_DEBUG
                /* 0 makes no problems with the next for loop */
                if( prop.multiProcessorCount <= 0 )
                {
                    std::cout << "[Warning] imresh::io::fillStreamList(): Devices has no multiprocessors. Ignoring this device." << std::endl;
                }
#           endif

            for( int j = 0; j < prop.multiProcessorCount; j++ )
            {
                stream str;
                str.device = i;
                CUDA_ERROR( hipStreamCreate( &str.str ) );
                streamList.push_back( str );
#               ifdef IMRESH_DEBUG
                    std::cout << "imresh::io::fillStreamList(): Created stream "
                        << j << " on device " << i << std::endl;
#               endif
            }
        }
#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::fillStreamList(): Finished stream creation."
                << std::endl;
#       endif

        return streamList.size( );
    }

    void taskQueueInit( )
    {
        threadPoolMaxSize = fillStreamList( );
#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::taskQueueInit(): Finished initilization."
                << std::endl;
#       endif
    }

    void taskQueueDeinit( )
    {
        threadPoolMaxSize = 0;

        while( threadPool.size( ) > 0 )
        {
            threadPool.front( ).join( );
            threadPool.pop_front( );
        }

        while( streamList.size( ) > 0 )
        {
            CUDA_ERROR( hipStreamDestroy( streamList.front( ).str ) );
            streamList.pop_front( );
        }

#       ifdef IMRESH_DEBUG
            std::cout << "imresh::io::taskQueueDeinit(): Finished deinitilization."
                << std::endl;
#       endif
    }

} // namespace io
} // namespace imresh
