#include "hip/hip_runtime.h"
/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2015-2016 Maximilian Knespel, Phillip Trommler
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include "cudaShrinkWrap.h"

#ifndef NDEBUG
#   define DEBUG_CUDASHRINKWRAP 0  // change this if you want to turn on debugging
#else
#   define DEBUG_CUDASHRINKWRAP 2  // leave this as it is
#endif

#include <iostream>
#include <cstddef>      // NULL
#include <cstring>      // memcpy
#include <cassert>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>       // atomicCAS
#include <hipfft/hipfft.h>
#include <utility>      // std::pair
#include "algorithms/cuda/cudaGaussian.h"
#include "algorithms/cuda/cudaVectorReduce.hpp"
#if DEBUG_CUDASHRINKWRAP == 1
#    include <fftw3.h>    // kinda problematic to mix this with cufft, but should work if it isn't hipfft/hipfftw.h
#    include "algorithms/vectorReduce.hpp"
#    include "algorithms/vectorElementwise.hpp"
#endif
#if DEBUG_CUDASHRINKWRAP == 2
#   include "io/writeOutFuncs/writeOutFuncs.hpp"
#endif
#include "libs/cudacommon.hpp"
#include "libs/checkCufftError.hpp"
#include "cudaVectorElementwise.hpp"


namespace imresh
{
namespace algorithms
{
namespace cuda
{


    template< class T_COMPLEX, class T_PREC >
    __global__ void cudaKernelApplyHioDomainConstraints
    (
        T_COMPLEX       * const __restrict__ rdpgPrevious,
        T_COMPLEX const * const __restrict__ rdpgPrime,
        T_PREC    const * const __restrict__ rdpIsMasked,
        unsigned int const rnElements,
        T_PREC const rHioBeta
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        int i = blockIdx.x * blockDim.x + threadIdx.x;
        const int nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            if ( rdpIsMasked[i] == 1 or /* g' */ rdpgPrime[i].x < 0 )
            {
                rdpgPrevious[i].x -= rHioBeta * rdpgPrime[i].x;
                rdpgPrevious[i].y -= rHioBeta * rdpgPrime[i].y;
            }
            else
            {
                rdpgPrevious[i].x = rdpgPrime[i].x;
                rdpgPrevious[i].y = rdpgPrime[i].y;
            }
        }
    }


#   ifdef IMRESH_DEBUG
        /**
         * checks if the imaginary parts are all 0. For debugging purposes
         **/
        template< class T_COMPLEX >
        void checkIfReal
        (
            const T_COMPLEX * const & rData,
            const unsigned & rnElements
        )
        {
            float avgRe = 0;
            float avgIm = 0;

            for ( unsigned i = 0; i < rnElements; ++i )
            {
                avgRe += fabs( rData[i][0] );
                avgIm += fabs( rData[i][1] );
            }

            avgRe /= (float) rnElements;
            avgIm /= (float) rnElements;

            std::cout << std::scientific
                      << "Avg. Re = " << avgRe << "\n"
                      << "Avg. Im = " << avgIm << "\n";
            assert( avgIm <  1e-5 );
        }
#   endif


    template< class T_PREC >
    float compareCpuWithGpuArray
    (
        T_PREC const * const __restrict__ rpData,
        T_PREC const * const __restrict__ rdpData,
        unsigned int const rnElements
    )
    {
        /* copy data from GPU in order to compare it */
        const unsigned nBytes = rnElements * sizeof(T_PREC);
        const T_PREC * const vec1 = rpData;
        T_PREC * const vec2 = (T_PREC*) malloc( nBytes );
        CUDA_ERROR( hipMemcpy( (void*) vec2, (void*) rdpData, nBytes, hipMemcpyDeviceToHost ) );

        float relErr = 0;

        //#pragma omp parallel for reduction( + : relErr )
        for ( unsigned i = 0; i < rnElements; ++i )
        {
            float max = fmax( fabs(vec1[i]), fabs(vec2[i]) );
            /* ignore 0/0 if both are equal and 0 */
            if ( max == 0 )
                max = 1;
            relErr += fabs( vec1[i] - vec2[i] ); // / max;
            //if ( i < 10 )
            //    std::cout << "    " << vec1[i] << " <-> " << vec2[i] << "\n";
        }

        free( vec2 );
        return relErr / rnElements;
    }


    int cudaShrinkWrap
    (
        float * const rIntensity,
        unsigned const rImageWidth,
        unsigned const rImageHeight,
        hipStream_t const rStream,
        unsigned int nBlocks,
        unsigned int nThreads,
        unsigned int rnCycles,
        float rTargetError,
        float rHioBeta,
        float rIntensityCutOffAutoCorel,
        float rIntensityCutOff,
        float rSigma0,
        float rSigmaChange,
        unsigned int rnHioCycles
    )
    {
        /* load libraries and functions which we need */
        using namespace imresh::algorithms;

        /* Evaluate input parameters and fill with default values if necessary */
        assert( rImageWidth > 0 );
        assert( rImageHeight > 0 );
        assert( rIntensity != NULL );
        /* this makes it possible to specifiy new values for e.g. rSigma0,
         * while still using the default values for rHioBeta, rTargetError,
         * ... */
        if ( rTargetError              <= 0 ) rTargetError              = 1e-5;
        if ( rnHioCycles               == 0 ) rnHioCycles               = 20;
        if ( rHioBeta                  <= 0 ) rHioBeta                  = 0.9;
        if ( rIntensityCutOffAutoCorel <= 0 ) rIntensityCutOffAutoCorel = 0.04;
        if ( rIntensityCutOff          <= 0 ) rIntensityCutOff          = 0.2;
        if ( rSigma0                   <= 0 ) rSigma0                   = 3.0;
        if ( rSigmaChange              <= 0 ) rSigmaChange              = 0.01;

        float sigma = rSigma0;
        unsigned const nElements = rImageWidth * rImageHeight;

        /* allocate needed memory so that HIO doesn't need to allocate and
         * deallocate on each call */
        hipfftComplex * dpCurData;
        hipfftComplex * dpgPrevious;
        float * dpIntensity;
        float * dpIsMasked;
        CUDA_ERROR( hipMalloc( (void**)&dpCurData  , sizeof(dpCurData  [0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpgPrevious, sizeof(dpgPrevious[0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpIntensity, sizeof(dpIntensity[0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpIsMasked , sizeof(dpIsMasked [0])*nElements ) );
        CUDA_ERROR( hipMemcpyAsync( dpIntensity, rIntensity, sizeof(dpIntensity[0])*nElements, hipMemcpyHostToDevice, rStream ) );

        /* create fft plans G' to g' and g to G */
        hipfftHandle ftPlan;
        CUFFT_ERROR( hipfftPlan2d( &ftPlan, rImageHeight /* nRows */, rImageWidth /* nColumns */, HIPFFT_C2C ) );
        CUFFT_ERROR( hipfftSetStream( ftPlan, rStream ) );

        /* create first guess for mask from autocorrelation (fourier transform
         * of the intensity @see
         * https://en.wikipedia.org/wiki/Wiener%E2%80%93Khinchin_theorem */
        cudaKernelCopyToRealPart<<<nBlocks,nThreads,0,rStream >>>( dpCurData, dpIntensity, nElements );
        CUDA_ERROR( hipPeekAtLastError() );

        CUFFT_ERROR( hipfftExecC2C( ftPlan, dpCurData, dpCurData, HIPFFT_BACKWARD ) );
        cudaKernelComplexNormElementwise<<<nBlocks,nThreads,0,rStream >>>( dpIsMasked, dpCurData, nElements );
        CUDA_ERROR( hipPeekAtLastError() );
        cudaGaussianBlur( dpIsMasked, rImageWidth, rImageHeight, sigma, rStream,
                          true /* don't call hipDeviceSynchronize */ );

        /* apply threshold to make binary mask */
        const float maskedAbsMax = cudaVectorMax( dpIsMasked, nElements, rStream );
        const float maskedThreshold = rIntensityCutOffAutoCorel * maskedAbsMax;
        cudaKernelCutOff<<<nBlocks,nThreads,0,rStream>>>( dpIsMasked, nElements, maskedThreshold, 1.0f, 0.0f );
        CUDA_ERROR( hipPeekAtLastError() );

        /* copy original image into fftw_complex array @todo: add random phase */
        cudaKernelCopyToRealPart<<<nBlocks,nThreads,0,rStream>>>( dpCurData, dpIntensity, nElements );
        CUDA_ERROR( hipPeekAtLastError() );

        /* in the first step the last value for g is to be approximated
         * by g'. The last value for g, called g_k is needed, because
         * g_{k+1} = g_k - hioBeta * g' ! This is inside the loop
         * because the fft is needed */
        hipMemcpyAsync( dpgPrevious, dpCurData, sizeof(dpCurData[0]) * nElements,
                    hipMemcpyDeviceToDevice, rStream );

        /* repeatedly call HIO algorithm and change mask */
        for ( unsigned iCycleShrinkWrap = 0; iCycleShrinkWrap < rnCycles; ++iCycleShrinkWrap )
        {
            /************************** Update Mask ***************************/

            /* blur |g'| (normally g' should be real!, so |.| not necessary) */
            cudaKernelComplexNormElementwise<<<nBlocks,nThreads,0,rStream>>>( dpIsMasked, dpCurData, nElements );
            CUDA_ERROR( hipPeekAtLastError() );
            cudaGaussianBlur( dpIsMasked, rImageWidth, rImageHeight, sigma, rStream, true /* don't call hipDeviceSynchronize */ );

            /* apply threshold to make binary mask */
            const float absMax = cudaVectorMax( dpIsMasked, nElements, rStream );
            const float threshold = rIntensityCutOff * absMax;
            cudaKernelCutOff<<<nBlocks,nThreads,0,rStream>>>( dpIsMasked, nElements, threshold, 1.0f, 0.0f );
            CUDA_ERROR( hipPeekAtLastError() );

            /* update the blurring sigma */
            sigma = fmax( 1.5f, ( 1.0f - rSigmaChange ) * sigma );

            for ( unsigned iHioCycle = 0; iHioCycle < rnHioCycles; ++iHioCycle )
            {
                /* apply domain constraints to g' to get g */
                cudaKernelApplyHioDomainConstraints<<<nBlocks,nThreads,0,rStream >>>
                    ( dpgPrevious, dpCurData, dpIsMasked, nElements, rHioBeta );
                CUDA_ERROR( hipPeekAtLastError() );

                /* Transform new guess g for f back into frequency space G' */
                CUFFT_ERROR( hipfftExecC2C( ftPlan, dpgPrevious, dpCurData, HIPFFT_FORWARD ) );

                /* Replace absolute of G' with measured absolute |F| */
                cudaKernelApplyComplexModulus<<<nBlocks,nThreads,0,rStream>>>
                    ( dpCurData, dpCurData, dpIntensity, nElements );
                CUDA_ERROR( hipPeekAtLastError() );

                CUFFT_ERROR( hipfftExecC2C( ftPlan, dpCurData, dpCurData, HIPFFT_BACKWARD ) );
            } // HIO loop

            /* check if we are done */
            const float currentError = cudaCalculateHioError( dpCurData /*g'*/, dpIsMasked, nElements, false /* don't invert mask */, rStream );
#           ifdef IMRESH_DEBUG
                std::cout << "[Error " << currentError << "/" << rTargetError << "] "
                          << "[Cycle " << iCycleShrinkWrap << "/" << rnCycles-1 << "]"
                          << std::endl;
#           endif
            if ( rTargetError > 0 && currentError < rTargetError )
                break;
            if ( iCycleShrinkWrap >= rnCycles )
                break;
        } // shrink wrap loop
        cudaKernelCopyFromRealPart<<<nBlocks,nThreads,0,rStream>>>( dpIntensity, dpCurData, nElements );
        CUDA_ERROR( hipPeekAtLastError() );
        CUDA_ERROR( hipMemcpyAsync( rIntensity, dpIntensity, sizeof(rIntensity[0])*nElements, hipMemcpyDeviceToHost, rStream ) );

        /* wait for everything to finish */
        CUDA_ERROR( hipStreamSynchronize( rStream ) );

        /* free buffers and plans */
        CUFFT_ERROR( hipfftDestroy( ftPlan ) );
        CUDA_ERROR( hipFree( dpCurData   ) );
        CUDA_ERROR( hipFree( dpgPrevious ) );
        CUDA_ERROR( hipFree( dpIntensity ) );
        CUDA_ERROR( hipFree( dpIsMasked  ) );

        return 0;
    }


} // namespace cuda
} // namespace algorithms
} // namespace imresh
