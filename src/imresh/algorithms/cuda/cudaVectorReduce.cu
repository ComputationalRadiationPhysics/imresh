#include "hip/hip_runtime.h"
/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2015-2016 Maximilian Knespel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */


#include "cudaVectorReduce.hpp"
#include "libs/cudacommon.h"


namespace imresh
{
namespace algorithms
{
namespace cuda
{


    /**
     * simple functors to just get the sum of two numbers. To be used
     * for the binary vectorReduce function to make it a vectorSum or
     * vectorMin or vectorMax
     **/
    template<class T> struct SumFunctor {
        __device__ __host__ T operator() ( const T & a, const T & b )
        { return a+b; }
    };
    template<class T> struct MinFunctor {
        __device__ __host__ T operator() ( const T & a, const T & b )
        { if (a<b) return a; else return b; }
    };
    template<class T> struct MaxFunctor {
        __device__ __host__ T operator() ( const T & a, const T & b )
        { if (a>b) return a; else return b; }
    };


    template<class T_PREC, class T_FUNC>
    __device__ T_PREC atomicFunc
    (
        T_PREC * const rdpTarget,
        const T_PREC rValue,
        T_FUNC f
    )
    {
        /* atomicCAS only is defined for int and long long int, thats why we
         * need these roundabout casts */
        int assumed;
        int old = * (int*) rdpTarget;

        /* atomicCAS returns the value with which the current value 'assumed'
         * was compared. If the value changed between reading out to assumed
         * and calculating the reduced value and storing it back, then we
         * need to call this function again. (I hope the GPU has some
         * functionality to prevent synchronized i.e. neverending races ... */
        do
        {
            assumed = old;
            old = atomicCAS( (int*) rdpTarget, assumed,
                __float_as_int( f( __int_as_float(assumed), rValue ) ) );
        } while ( assumed != old );

        return __int_as_float( old );
    }


    template<class T_PREC, class T_FUNC>
    __global__ void kernelVectorReduceShared
    (
        const T_PREC * const rdpData,
        const unsigned rnData,
        T_PREC * const rdpResult,
        T_FUNC f,
        const T_PREC rInitValue
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        const int64_t nTotalThreads = gridDim.x * blockDim.x;
        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        assert( i < nTotalThreads );

        T_PREC localReduced = T_PREC(rInitValue);
        for ( ; i < rnData; i += nTotalThreads )
            localReduced = f( localReduced, rdpData[i] );

        __shared__ T_PREC smReduced;
        /* master thread of every block shall set shared mem variable to 0 */
        __syncthreads();
        if ( threadIdx.x == 0 )
            smReduced = T_PREC(rInitValue);
        __syncthreads();

        atomicFunc( &smReduced, localReduced, f );

        __syncthreads();
        if ( threadIdx.x == 0 )
            atomicFunc( rdpResult, smReduced, f );
    }


    /**
     * benchmarks suggest that this kernel is twice as fast as
     * kernelVectorReduceShared
     **/
    template<class T_PREC, class T_FUNC>
    __global__ void kernelVectorReduceSharedMemoryWarps
    (
        const T_PREC * const rdpData,
        const unsigned rnData,
        T_PREC * const rdpResult,
        T_FUNC f,
        const T_PREC rInitValue
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        const int64_t nTotalThreads = gridDim.x * blockDim.x;
        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        assert( i < nTotalThreads );

        T_PREC localReduced = T_PREC(rInitValue);
        for ( ; i < rnData; i += nTotalThreads )
            localReduced = f( localReduced, rdpData[i] );

        /**
         * reduce per warp:
         * With __shfl_down we can read the register values of other lanes in
         * a warp. In the first iteration lane 0 will add to it's value the
         * value of lane 16, lane 1 from lane 17 and so in.
         * In the next step lane 0 will add the result from lane 8.
         * In the end lane 0 will have the reduced value.
         * @see http://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
         **/
        constexpr int warpSize = 32;
        const int64_t laneId = threadIdx.x % warpSize;
        for ( int64_t warpDelta = warpSize / 2; warpDelta > 0; warpDelta /= 2)
            localReduced = f( localReduced, __shfl_down( localReduced, warpDelta ) );

        __shared__ T_PREC smReduced;
        /* master thread of every block shall set shared mem variable to 0 */
        __syncthreads();
        if ( threadIdx.x == 0 )
            smReduced = T_PREC(rInitValue);
        __syncthreads();

        if ( laneId == 0 )
            atomicFunc( &smReduced, localReduced, f );

        __syncthreads();
        if ( threadIdx.x == 0 )
            atomicFunc( rdpResult, smReduced, f );
    }


    template<class T_PREC, class T_FUNC>
    __global__ void kernelVectorReduceWarps
    (
        const T_PREC * const rdpData,
        const unsigned rnData,
        T_PREC * const rdpResult,
        T_FUNC f,
        const T_PREC rInitValue
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        const int64_t nTotalThreads = gridDim.x * blockDim.x;
        int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        assert( i < nTotalThreads );

        T_PREC localReduced = T_PREC(rInitValue);
        for ( ; i < rnData; i += nTotalThreads )
            localReduced = f( localReduced, rdpData[i] );

        /* reduce per warp */
        constexpr int warpSize = 32;
        const int64_t laneId = threadIdx.x % warpSize;
        for ( int64_t warpDelta = warpSize / 2; warpDelta > 0; warpDelta /= 2)
            localReduced = f( localReduced, __shfl_down( localReduced, warpDelta ) );

        if ( laneId == 0 )
            atomicFunc( rdpResult, localReduced, f );
    }


    template<class T_PREC, class T_FUNC>
    T_PREC cudaReduce
    (
        const T_PREC * const rdpData,
        const unsigned rnElements,
        T_FUNC f,
        const T_PREC rInitValue
    )
    {
        const unsigned nThreads = 256;
        //const unsigned nBlocks  = ceil( (float) rnElements / nThreads );
        //printf( "nThreads = %i, nBlocks = %i\n", nThreads, nBlocks );
        const unsigned nBlocks = 256;
        /* 256*256 = 65536 concurrent threads should fill most modern graphic
         * cards. E.g. GTX 760 can only handle 12288 runnin concurrently,
         * everything else will be run after some threads finished. The
         * number of kernels is only 384, because of oversubscription with
         * warps */
        assert( nBlocks < 65536 );

        T_PREC reducedValue;
        T_PREC * dpReducedValue;
        T_PREC initValue = rInitValue;

        CUDA_ERROR( hipMalloc( (void**) &dpReducedValue, sizeof(float) ) );
        CUDA_ERROR( hipMemcpy( dpReducedValue, &initValue, sizeof(float), hipMemcpyHostToDevice ) );

        kernelVectorReduceWarps<<<nBlocks,nThreads>>>
            ( rdpData, rnElements, dpReducedValue, f, rInitValue );

        CUDA_ERROR( hipDeviceSynchronize() );
        CUDA_ERROR( hipMemcpy( &reducedValue, dpReducedValue, sizeof(float), hipMemcpyDeviceToHost ) );

        CUDA_ERROR( hipFree( dpReducedValue ) );

        return reducedValue;
    }


    template<class T_PREC, class T_FUNC>
    T_PREC cudaReduceSharedMemory
    (
        const T_PREC * const rdpData,
        const unsigned rnElements,
        T_FUNC f,
        const T_PREC rInitValue
    )
    {
        /* the more threads we have the longer the reduction will be
         * done inside shared memory instead of global memory */
        const unsigned nThreads = 256;
        const unsigned nBlocks = 256;
        assert( nBlocks < 65536 );

        T_PREC reducedValue;
        T_PREC * dpReducedValue;
        T_PREC initValue = rInitValue;

        CUDA_ERROR( hipMalloc( (void**) &dpReducedValue, sizeof(float) ) );
        CUDA_ERROR( hipMemcpy( dpReducedValue, &initValue, sizeof(float), hipMemcpyHostToDevice ) );

        kernelVectorReduceShared<<<nBlocks,nThreads>>>
            ( rdpData, rnElements, dpReducedValue, f, rInitValue );

        CUDA_ERROR( hipDeviceSynchronize() );
        CUDA_ERROR( hipMemcpy( &reducedValue, dpReducedValue, sizeof(float), hipMemcpyDeviceToHost ) );

        CUDA_ERROR( hipFree( dpReducedValue ) );

        return reducedValue;
    }


    template<class T_PREC, class T_FUNC>
    T_PREC cudaReduceSharedMemoryWarps
    (
        const T_PREC * const rdpData,
        const unsigned rnElements,
        T_FUNC f,
        const T_PREC rInitValue
    )
    {
        const unsigned nThreads = 256;
        const unsigned nBlocks = 256;
        assert( nBlocks < 65536 );

        T_PREC reducedValue;
        T_PREC * dpReducedValue;
        T_PREC initValue = rInitValue;

        CUDA_ERROR( hipMalloc( (void**) &dpReducedValue, sizeof(float) ) );
        CUDA_ERROR( hipMemcpy( dpReducedValue, &initValue, sizeof(float), hipMemcpyHostToDevice ) );

        kernelVectorReduceSharedMemoryWarps<<<nBlocks,nThreads>>>
            ( rdpData, rnElements, dpReducedValue, f, rInitValue );

        CUDA_ERROR( hipDeviceSynchronize() );
        CUDA_ERROR( hipMemcpy( &reducedValue, dpReducedValue, sizeof(float), hipMemcpyDeviceToHost ) );

        CUDA_ERROR( hipFree( dpReducedValue ) );

        return reducedValue;
    }


    template<class T_PREC>
    T_PREC cudaVectorMin
    (
        const T_PREC * const rdpData,
        const unsigned rnElements
    )
    {
        MinFunctor<T_PREC> minFunctor;
        return cudaReduce( rdpData, rnElements, minFunctor, std::numeric_limits<T_PREC>::max() );
    }


    template<class T_PREC>
    T_PREC cudaVectorMax
    (
        const T_PREC * const rdpData,
        const unsigned rnElements
    )
    {
        MaxFunctor<T_PREC> maxFunctor;
        return cudaReduce( rdpData, rnElements, maxFunctor, std::numeric_limits<T_PREC>::lowest() );
    }


    template<class T_PREC>
    T_PREC cudaVectorSum
    (
        const T_PREC * const rdpData,
        const unsigned rnElements
    )
    {
        SumFunctor<T_PREC> sumFunctor;
        return cudaReduce( rdpData, rnElements, sumFunctor, T_PREC(0) );
    }


    /* These functions only persist for benchmarking purposes to show that
     * the standard version is the fastest */

    template<class T_PREC>
    T_PREC cudaVectorMaxSharedMemory
    (
        const T_PREC * const rdpData,
        const unsigned rnElements
    )
    {
        MaxFunctor<T_PREC> maxFunctor;
        return cudaReduceSharedMemory( rdpData, rnElements, maxFunctor, std::numeric_limits<T_PREC>::lowest() );
    }

    template<class T_PREC>
    T_PREC cudaVectorMaxSharedMemoryWarps
    (
        const T_PREC * const rdpData,
        const unsigned rnElements
    )
    {
        MaxFunctor<T_PREC> maxFunctor;
        return cudaReduceSharedMemoryWarps( rdpData, rnElements, maxFunctor, std::numeric_limits<T_PREC>::lowest() );
    }


    /**
     * "For the input-output algorithms the error E_F is
     *  usually meaningless since the input g_k(X) is no longer
     *  an estimate of the object. Then the meaningful error
     *  is the object-domain error E_0 given by Eq. (15)."
     *                                      (Fienup82)
     * Eq.15:
     * @f[ E_{0k}^2 = \sum\limits_{x\in\gamma} |g_k'(x)^2|^2 @f]
     * where \gamma is the domain at which the constraints are
     * not met. SO this is the sum over the domain which should
     * be 0.
     *
     * Eq.16:
     * @f[ E_{Fk}^2 = \sum\limits_{u} |G_k(u) - G_k'(u)|^2 / N^2
                    = \sum_x |g_k(x) - g_k'(x)|^2 @f]
     **/
    template< class T_COMPLEX, class T_MASK_ELEMENT >
    __global__ void cudaKernelCalculateHioError
    (
        const T_COMPLEX * const rdpgPrime,
        const T_MASK_ELEMENT * const rdpIsMasked,
        const unsigned rnData,
        const bool rInvertMask,
        float * const rdpTotalError,
        float * const rdpnMaskedPixels
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        assert( i < nTotalThreads );

        float localTotalError    = 0;
        float localnMaskedPixels = 0;
        for ( ; i < rnData; i += nTotalThreads )
        {
            const auto & re = rdpgPrime[i].x;
            const auto & im = rdpgPrime[i].y;

            /* only add up norm where no object should be (rMask == 0) */
            assert( rdpIsMasked[i] >= 0.0 and rdpIsMasked[i] <= 1.0 );
            float shouldBeZero = rdpIsMasked[i];
            if ( rInvertMask )
                shouldBeZero = 1 - shouldBeZero;

            localTotalError    += shouldBeZero * ( re*re+im*im );
            localnMaskedPixels += shouldBeZero;
        }

        __shared__ float smTotalError, smnMaskedPixels;
        /* master thread of every block shall set shared mem variable to 0 */
        __syncthreads();
        if ( threadIdx.x == 0 )
        {
            smTotalError    = 0;
            smnMaskedPixels = 0;
        }
        __syncthreads();

        SumFunctor<float> sum;
        atomicFunc( &smTotalError   , localTotalError   , sum );
        atomicFunc( &smnMaskedPixels, localnMaskedPixels, sum );

        __syncthreads();
        if ( threadIdx.x == 0 )
        {
            atomicFunc( rdpTotalError, smTotalError, sum );
            atomicFunc( rdpnMaskedPixels, smnMaskedPixels, sum );
        }
    }


    template<class T_COMPLEX, class T_MASK_ELEMENT>
    float calculateHioError
    (
        const T_COMPLEX * const & rdpData,
        const T_MASK_ELEMENT * const & rdpIsMasked,
        const unsigned & rnElements,
        const bool & rInvertMask
    )
    {
        const unsigned nThreads = 256;
        //const unsigned nBlocks  = ceil( (float) rnElements / nThreads );
        const unsigned nBlocks  = 256;
        assert( nBlocks < 65536 );

        float     totalError,     nMaskedPixels;
        float * dpTotalError, * dpnMaskedPixels;

        CUDA_ERROR( hipMalloc( (void**) &dpTotalError   , sizeof(float) ) );
        CUDA_ERROR( hipMalloc( (void**) &dpnMaskedPixels, sizeof(float) ) );
        CUDA_ERROR( hipMemset( dpTotalError   , 0, sizeof(float) ) );
        CUDA_ERROR( hipMemset( dpnMaskedPixels, 0, sizeof(float) ) );

        cudaKernelCalculateHioError<<<nBlocks,nThreads>>>
            ( rdpData, rdpIsMasked, rnElements, rInvertMask, dpTotalError, dpnMaskedPixels );
        CUDA_ERROR( hipDeviceSynchronize() );

        CUDA_ERROR( hipMemcpy( &totalError, dpTotalError, sizeof(float), hipMemcpyDeviceToHost ) );
        CUDA_ERROR( hipMemcpy( &nMaskedPixels, dpnMaskedPixels, sizeof(float), hipMemcpyDeviceToHost ) );

        CUDA_ERROR( hipFree( dpTotalError    ) );
        CUDA_ERROR( hipFree( dpnMaskedPixels ) );

        return sqrtf(totalError) / nMaskedPixels;
    }


    /* explicit instantiations */

    template
    float cudaVectorMin<float>
    (
        const float * const rdpData,
        const unsigned rnElements
    );
    template
    double cudaVectorMin<double>
    (
        const double * const rdpData,
        const unsigned rnElements
    );


    template
    float cudaVectorMax<float>
    (
        const float * const rdpData,
        const unsigned rnElements
    );
    template
    double cudaVectorMax<double>
    (
        const double * const rdpData,
        const unsigned rnElements
    );


    template
    float cudaVectorSum<float>
    (
        const float * const rdpData,
        const unsigned rnElements
    );
    template
    double cudaVectorSum<double>
    (
        const double * const rdpData,
        const unsigned rnElements
    );

    template
    __global__ void cudaKernelCalculateHioError
    <hipfftComplex, float>
    (
        const hipfftComplex * const rdpgPrime,
        const float * const rdpIsMasked,
        const unsigned rnData,
        const bool rInvertMask,
        float * const rdpTotalError,
        float * const rdpnMaskedPixels
    );


    template
    float calculateHioError
    <hipfftComplex, float>
    (
        const hipfftComplex * const & rdpData,
        const float * const & rdpIsMasked,
        const unsigned & rnElements,
        const bool & rInvertMask
    );


    template
    float cudaVectorMaxSharedMemory<float>
    (
        const float * const rdpData,
        const unsigned rnElements
    );

    template
    float cudaVectorMaxSharedMemoryWarps<float>
    (
        const float * const rdpData,
        const unsigned rnElements
    );


} // namespace cuda
} // namespace algorithms
} // namespace imresh
