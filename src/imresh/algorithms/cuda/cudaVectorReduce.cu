#include "hip/hip_runtime.h"
/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2015-2016 Maximilian Knespel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include "cudaVectorReduce.hpp"

#include <cassert>
#include <cstdio>
#include <cstdint>    // uint64_t
#include <limits>     // lowest
#include <cmath>
#include <hip/hip_runtime.h>     // atomicCAS
#include <hipfft/hipfft.h>    // hipfftComplex, hipfftDoubleComplex
#include "libs/cudacommon.hpp"


namespace imresh
{
namespace algorithms
{
namespace cuda
{


    SumFunctor<float > sumFunctorf;
    MinFunctor<float > minFunctorf;
    MaxFunctor<float > maxFunctorf;
    SumFunctor<double> sumFunctord;
    MinFunctor<double> minFunctord;
    MaxFunctor<double> maxFunctord;


    template<class T_FUNC>
    __device__ inline void atomicFunc
    (
        float * const rdpTarget,
        float const rValue,
        T_FUNC f
    )
    {
        /* atomicCAS only is defined for int and long long int, thats why we
         * need these roundabout casts */
        uint32_t assumed;
        uint32_t old = * (uint32_t*) rdpTarget;

        /* atomicCAS returns the value with which the current value 'assumed'
         * was compared. If the value changed between reading out to assumed
         * and calculating the reduced value and storing it back, then we
         * need to call this function again. (I hope the GPU has some
         * functionality to prevent synchronized i.e. neverending races ... */
        do
        {
            assumed = old;

            /* If the reduced value doesn't change, then we don't need to hinder
             * other threads with atomicCAS. This additional check may prove a
             * bottleneck, if this is rarely the case, e.g. for sum and no 0s or
             * for max and an ordered list, where the largest is the last
             * element. In tests this more often slowed down the calculation */
            //if ( f( __int_as_float(assumed), rValue ) == assumed )
            //    break;

            /* compare and swap after the value was read with assumend, return
             * old value, if assumed isn't anymore the value at rdpTarget,
             * then we will have to try again to write it */
            old = atomicCAS( (uint32_t*) rdpTarget, assumed,
                __float_as_int( f( __int_as_float(assumed), rValue ) ) );
        }
        while ( assumed != old );
    }

    template<class T_FUNC>
    __device__ inline void atomicFunc
    (
        double * const rdpTarget,
        double const rValue,
        T_FUNC f
    )
    {
        using ull = unsigned long long int;
        ull assumed;
        ull old = * (ull*) rdpTarget;
        do
        {
            assumed = old;
            old = atomicCAS( (ull*) rdpTarget, assumed,
                __double_as_longlong( f( __longlong_as_double(assumed), rValue ) ) );
        }
        while ( assumed != old );
    }


    template<>
    __device__ inline void atomicFunc<int,MaxFunctor<int>>
    (
        int * const rdpTarget,
        int const rValue,
        MaxFunctor<int> f
    )
    {
        atomicMax( rdpTarget, rValue );
    }


    /*
    // seems to work for testVectorReduce, but it shouldn't oO, maybe just good numbers, or because this is only for max, maybe it wouldn't work for min, because the maximum is > 0 ... In the end it isn't faster than atomicCAS and it doesn't even use floatAsOrderdInt yet, which would make use of bitshift, subtraction and logical or, thereby decreasing performance even more: http://stereopsis.com/radix.html
    template<>
    __device__ inline void atomicFunc<float,MaxFunctor<float>>
    (
        float * const rdpTarget,
        const float rValue,
        MaxFunctor<float> f
    )
    {
        atomicMax( (int*)rdpTarget, __float_as_int(rValue) );
    }*/

    template<class T_PREC, class T_FUNC>
    __global__ void kernelVectorReduce
    (
        T_PREC const * const __restrict__ rdpData,
        unsigned int const rnData,
        T_PREC * const __restrict__ rdpResult,
        T_FUNC f,
        T_PREC const rInitValue
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        auto iElem = rdpData + blockIdx.x * blockDim.x + threadIdx.x;
        auto localReduced = T_PREC( rInitValue );
        #pragma unroll
        for ( ; iElem < rdpData + rnData; iElem += gridDim.x * blockDim.x )
            localReduced = f( localReduced, *iElem );

        /* reduce per warp (warpSize == 32 assumed) */
        int constexpr cWarpSize = 32;
        assert( cWarpSize == warpSize );
        #pragma unroll
        for ( int32_t warpDelta = cWarpSize / 2; warpDelta > 0; warpDelta /= 2)
            localReduced = f( localReduced, __shfl_down( localReduced, warpDelta ) );

        if ( threadIdx.x % cWarpSize == 0 )
            atomicFunc( rdpResult, localReduced, f );
    }


    template<class T_PREC, class T_FUNC>
    T_PREC cudaReduce
    (
        T_PREC const * const rdpData,
        unsigned int const rnElements,
        T_FUNC f,
        T_PREC const rInitValue,
        hipStream_t rStream
    )
    {
        const unsigned nThreads = 128;
        //const unsigned nBlocks  = ceil( (float) rnElements / nThreads );
        //printf( "nThreads = %i, nBlocks = %i\n", nThreads, nBlocks );
        const unsigned nBlocks = 288;
        /* 256*256 = 65536 concurrent threads should fill most modern graphic
         * cards. E.g. GTX 760 can only handle 12288 runnin concurrently,
         * everything else will be run after some threads finished. The
         * number of kernels is only 384, because of oversubscription with
         * warps */
        assert( nBlocks < 65536 );

        T_PREC reducedValue;
        T_PREC * dpReducedValue;
        T_PREC initValue = rInitValue;

        CUDA_ERROR( hipMalloc( (void**) &dpReducedValue, sizeof(T_PREC) ) );
        CUDA_ERROR( hipMemcpyAsync( dpReducedValue, &initValue, sizeof(T_PREC),
                                     hipMemcpyHostToDevice, rStream ) );

        /* memcpy is on the same stream as kernel will be, so no synchronize needed! */
        kernelVectorReduce<<< nBlocks, nThreads, 0, rStream >>>
            ( rdpData, rnElements, dpReducedValue, f, rInitValue );
        CUDA_ERROR( hipPeekAtLastError() );

        CUDA_ERROR( hipStreamSynchronize( rStream ) );
        CUDA_ERROR( hipMemcpyAsync( &reducedValue, dpReducedValue, sizeof(T_PREC),
                                     hipMemcpyDeviceToHost, rStream ) );
        CUDA_ERROR( hipStreamSynchronize( rStream) );
        CUDA_ERROR( hipFree( dpReducedValue ) );

        return reducedValue;
    }

    template<class T_PREC>
    T_PREC cudaVectorMin
    (
        T_PREC const * const rdpData,
        unsigned int const rnElements,
        hipStream_t rStream
    )
    {
        MinFunctor<T_PREC> minFunctor;
        return cudaReduce( rdpData, rnElements, minFunctor, std::numeric_limits<T_PREC>::max(), rStream );
    }


    template<class T_PREC>
    T_PREC cudaVectorMax
    (
        T_PREC const * const rdpData,
        unsigned int const rnElements,
        hipStream_t rStream
    )
    {
        MaxFunctor<T_PREC> maxFunctor;
        return cudaReduce( rdpData, rnElements, maxFunctor, std::numeric_limits<T_PREC>::lowest(), rStream );
    }


    template<class T_PREC>
    T_PREC cudaVectorSum
    (
        T_PREC const * const rdpData,
        unsigned int const rnElements,
        hipStream_t rStream
    )
    {
        SumFunctor<T_PREC> sumFunctor;
        return cudaReduce( rdpData, rnElements, sumFunctor, T_PREC(0), rStream );
    }

    inline __device__ uint32_t getLaneId( void )
    {
        uint32_t id;
        asm("mov.u32 %0, %%laneid;" : "=r"(id));
        return id;
    }

    /**
     * "For the input-output algorithms the error @f[ E_F @f] is
     *  usually meaningless since the input @f[ g_k(X) @f] is no longer
     *  an estimate of the object. Then the meaningful error
     *  is the object-domain error @f[ E_0 @f] given by Eq. (15)."
     *                                      (Fienup82)
     * Eq.15:
     * @f[ E_{0k}^2 = \sum\limits_{x\in\gamma} |g_k'(x)^2|^2 @f]
     * where @f[ \gamma @f] is the domain at which the constraints are
     * not met. So this is the sum over the domain which should
     * be 0.
     *
     * Eq.16:
     * @f[ E_{Fk}^2 = \sum\limits_{u} |G_k(u) - G_k'(u)|^2 / N^2
     *              = \sum_x |g_k(x) - g_k'(x)|^2 @f]
     *
     * Note that all pointers may not overlap with each other!
     * Some possible restrictions on the gridSize and blockSize
     *   - Every thread should at least do SOME work for the overhead to
     *     amortize. I suspect that 32 elements per thread can be a good
     *     value, but if you can fill the GPU with some other task meanwhile
     *     you should go even higher.
     **/
    template< class T_COMPLEX, class T_MASK >
    __global__ void cudaKernelCalculateHioError
    (
        T_COMPLEX const * const __restrict__ rdpData,
        T_MASK    const * const __restrict__ rdpIsMasked,
        unsigned int const rnData,
        bool const rInvertMask,
        float * const __restrict__ rdpTotalError,
        float * const __restrict__ rdpnMaskedPixels
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        auto const nTotalThreads = gridDim.x * blockDim.x;
        auto iElem = rdpData     + blockIdx.x * blockDim.x + threadIdx.x;
        auto iMask = rdpIsMasked + blockIdx.x * blockDim.x + threadIdx.x;

        float localTotalError    = 0;
        float localnMaskedPixels = 0;
        #pragma unroll
        for ( ; iElem < rdpData + rnData; iElem += nTotalThreads, iMask += nTotalThreads )
        {
            auto const re = iElem->x;
            auto const im = iElem->y;

            /* only add up norm where no object should be (rMask == 0) */
            /* note: invert   + masked   -> unmasked  <=> 1 ? 1 -> 0
             *       noinvert + masked   -> masked    <=> 0 ? 1 -> 1
             *       invert   + unmasked -> masked    <=> 1 ? 0 -> 1
             *       noinvert + unmasked -> unmasked  <=> 0 ? 0 -> 0
             *   => ? is xor    => no thread divergence
             */
            #ifndef NDEBUG
                if ( not ( *iMask == 0 or *iMask == 1 ) )
                {
                    printf( "rdpIsMasked[%i] = %u\n", iMask-rdpIsMasked, *iMask );
                    assert( *iMask == 0 or *iMask == 1 );
                }
            #endif
            const bool shouldBeZero = rInvertMask xor (bool) *iMask;
            assert( *iMask >= 0.0 and *iMask <= 1.0 );
            //float shouldBeZero = rInvertMask + ( 1-2*rInvertMask )**iMask;
            /*
            float shouldBeZero = rdpIsMasked[i];
            if ( rInvertMask )
                shouldBeZero = 1 - shouldBeZero;
            */

            localTotalError    += shouldBeZero * sqrtf( re*re+im*im );
            localnMaskedPixels += shouldBeZero;
        }

        /* reduce per warp (warpSize == 32 assumed) */
        int constexpr cWarpSize = 32;
        assert( cWarpSize == warpSize );
        #pragma unroll
        for ( int32_t warpDelta = cWarpSize / 2; warpDelta > 0; warpDelta /= 2 )
        {
            localTotalError    += __shfl_down( localTotalError   , warpDelta );
            localnMaskedPixels += __shfl_down( localnMaskedPixels, warpDelta );
        }

        assert( getLaneId() == threadIdx.x % cWarpSize );
        if ( getLaneId() == 0 )
        {
            atomicAdd( rdpTotalError   , localTotalError    );
            atomicAdd( rdpnMaskedPixels, localnMaskedPixels );
        }
    }

    template<class T_COMPLEX, class T_MASK>
    float cudaCalculateHioError
    (
        T_COMPLEX const * const rdpData,
        T_MASK const * const rdpIsMasked,
        unsigned int const rnElements,
        bool const rInvertMask,
        hipStream_t rStream,
        float * const rpTotalError,
        float * const rpnMaskedPixels
    )
    {
        const unsigned nThreads = 256;
        //const unsigned nBlocks  = ceil( (float) rnElements / nThreads );
        const unsigned nBlocks  = 256;
        assert( nBlocks < 65536 );

        float     totalError,     nMaskedPixels;
        float * dpTotalError, * dpnMaskedPixels;

        CUDA_ERROR( hipMalloc( (void**) &dpTotalError   , sizeof(float) ) );
        CUDA_ERROR( hipMalloc( (void**) &dpnMaskedPixels, sizeof(float) ) );
        CUDA_ERROR( hipMemsetAsync( dpTotalError   , 0, sizeof(float), rStream ) );
        CUDA_ERROR( hipMemsetAsync( dpnMaskedPixels, 0, sizeof(float), rStream ) );

        /* memset is on the same stream as kernel will be, so no synchronize needed! */
        cudaKernelCalculateHioError<<< nBlocks, nThreads, 0, rStream >>>
            ( rdpData, rdpIsMasked, rnElements, rInvertMask, dpTotalError, dpnMaskedPixels );
        CUDA_ERROR( hipPeekAtLastError() );
        CUDA_ERROR( hipStreamSynchronize( rStream ) );

        CUDA_ERROR( hipMemcpyAsync( &totalError   , dpTotalError   , sizeof(float), hipMemcpyDeviceToHost, rStream ) );
        CUDA_ERROR( hipMemcpyAsync( &nMaskedPixels, dpnMaskedPixels, sizeof(float), hipMemcpyDeviceToHost, rStream ) );
        CUDA_ERROR( hipStreamSynchronize( rStream ) );

        CUDA_ERROR( hipFree( dpTotalError    ) );
        CUDA_ERROR( hipFree( dpnMaskedPixels ) );

        if ( rpTotalError != NULL )
            *rpTotalError    = totalError;
        if ( rpnMaskedPixels != NULL )
            *rpnMaskedPixels = nMaskedPixels;

        return sqrtf(totalError) / nMaskedPixels;
    }


    /* explicit template instantiations */

    template
    float cudaVectorMin<float>
    (
        float const * rdpData,
        unsigned int rnElements,
        hipStream_t rStream
    );
    template
    double cudaVectorMin<double>
    (
        double const * rdpData,
        unsigned int rnElements,
        hipStream_t rStream
    );


    template
    float cudaVectorMax<float>
    (
        float const * rdpData,
        unsigned int rnElements,
        hipStream_t rStream
    );
    template
    double cudaVectorMax<double>
    (
        double const * rdpData,
        unsigned int rnElements,
        hipStream_t rStream
    );


    template
    float cudaVectorSum<float>
    (
        float const * rdpData,
        unsigned int rnElements,
        hipStream_t rStream
    );
    template
    double cudaVectorSum<double>
    (
        double const * rdpData,
        unsigned int rnElements,
        hipStream_t rStream
    );

    template
    __global__ void cudaKernelCalculateHioError
    <hipfftComplex, float>
    (
        hipfftComplex const * rdpgPrime,
        float const * rdpIsMasked,
        unsigned int rnData,
        bool rInvertMask,
        float * rdpTotalError,
        float * rdpnMaskedPixels
    );


    template
    float cudaCalculateHioError
    <hipfftComplex, float>
    (
        hipfftComplex const * rdpData,
        float const * rdpIsMasked,
        unsigned int rnElements,
        bool rInvertMask,
        hipStream_t rStream,
        float * rdpTotalError,
        float * rdpnMaskedPixels
    );
    template
    float cudaCalculateHioError
    <hipfftComplex, bool>
    (
        hipfftComplex const * rdpData,
        bool const * rdpIsMasked,
        unsigned int rnElements,
        bool rInvertMask,
        hipStream_t rStream,
        float * rdpTotalError,
        float * rdpnMaskedPixels
    );
    template
    float cudaCalculateHioError
    <hipfftComplex, unsigned char>
    (
        hipfftComplex const * rdpData,
        unsigned char const * rdpIsMasked,
        unsigned int rnElements,
        bool rInvertMask,
        hipStream_t rStream,
        float * rdpTotalError,
        float * rdpnMaskedPixels
    );


} // namespace cuda
} // namespace algorithms
} // namespace imresh
