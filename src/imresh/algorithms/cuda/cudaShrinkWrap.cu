#include "hip/hip_runtime.h"
/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2015-2016 Maximilian Knespel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <utility>                              // std::pair

#include "algorithms/cuda/cudaShrinkWrap.h"
#include "libs/cudacommon.h"


namespace imresh
{
namespace algorithms
{
namespace cuda
{


    template< class T_COMPLEX, class T_PREC >
    __global__ void cudaKernelCopyToRealPart
    (
        T_COMPLEX * const rTargetComplexArray,
        T_PREC    * const rSourceRealArray,
        unsigned    const rnElements
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            rTargetComplexArray[i].x = rSourceRealArray[i]; /* Re */
            rTargetComplexArray[i].y = 0;
        }
    }


    template< class T_PREC, class T_COMPLEX >
    __global__ void cudaKernelCopyFromRealPart
    (
        T_PREC    * const rTargetComplexArray,
        T_COMPLEX * const rSourceRealArray,
        unsigned    const rnElements
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            rTargetComplexArray[i] = rSourceRealArray[i].x; /* Re */
        }
    }


    template< class T_PREC, class T_COMPLEX >
    __global__ void cudaKernelComplexNormElementwise
    (
        T_PREC * const rdpDataTarget,
        const T_COMPLEX * const rdpDataSource,
        const unsigned rnElements
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            const float & re = rdpDataSource[i].x;
            const float & im = rdpDataSource[i].y;
            rdpDataTarget[i] = sqrtf( re*re + im*im );
        }
    }


    template< class T_COMPLEX, class T_PREC >
    __global__ void cudaKernelApplyComplexModulus
    (
        T_COMPLEX * const rdpDataTarget,
        const T_COMPLEX * const rdpDataSource,
        const T_PREC * const rdpComplexModulus,
        const unsigned rnElements
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            const auto & re = rdpDataSource[i].x;
            const auto & im = rdpDataSource[i].y;
            auto norm = sqrtf(re*re+im*im);
            if ( norm == 0 ) // in order to avoid NaN
                norm = 1;
            const float factor = rdpComplexModulus[i] / norm;
            rdpDataTarget[i].x = re * factor;
            rdpDataTarget[i].y = im * factor;
        }
    }


    template< class T_PREC >
    __global__ void cudaKernelCutOff
    (
        T_PREC * const rData,
        unsigned const rnElements,
        const T_PREC rThreshold,
        const T_PREC rLowerValue,
        const T_PREC rUpperValue
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            rData[i] = rData[i] < rThreshold ? rLowerValue : rUpperValue;
        }
    }


    template< class T_COMPLEX, class T_PREC >
    __global__ void cudaKernelApplyHioDomainConstraints
    (
        T_COMPLEX * const rdpgPrevious,
        const T_COMPLEX * const rdpgPrime,
        const T_PREC * const rdpIsMasked,
        unsigned const rnElements,
        const T_PREC rHioBeta
    )
    {
        assert( blockDim.y == 1 );
        assert( blockDim.z == 1 );
        assert( gridDim.y  == 1 );
        assert( gridDim.z  == 1 );

        uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t nTotalThreads = gridDim.x * blockDim.x;
        for ( ; i < rnElements; i += nTotalThreads )
        {
            if ( rdpIsMasked[i] == 1 or /* g' */ rdpgPrime[i].x < 0 )
            {
                rdpgPrevious[i].x -= rHioBeta * rdpgPrime[i].x;
                rdpgPrevious[i].y -= rHioBeta * rdpgPrime[i].y;
            }
            else
            {
                rdpgPrevious[i].x = rdpgPrime[i].x;
                rdpgPrevious[i].y = rdpgPrime[i].y;
            }
        }
    }


    /**
     * Shifts the Fourier transform result in frequency space to the center
     *
     * @verbatim
     *        +------------+      +------------+          +------------+
     *        |            |      |78 ++  ++ 56|          |     --     |
     *        |            |      |o> ''  '' <o|          | .. <oo> .. |
     *        |     #      |  FT  |-          -| fftshift | ++ 1234 ++ |
     *        |     #      |  ->  |-          -|  ----->  | ++ 5678 ++ |
     *        |            |      |o> ..  .. <o|          | '' <oo> '' |
     *        |            |      |34 ++  ++ 12|          |     --     |
     *        +------------+      +------------+          +------------+
     *                           k=0         k=N-1              k=0
     * @endverbatim
     * This index shift can be done by a simple shift followed by a modulo:
     *   newArray[i] = array[ (i+N/2)%N ]
     **/
    template< class T_COMPLEX >
    void fftShift
    (
        T_COMPLEX * const & data,
        const unsigned & Nx,
        const unsigned & Ny
    )
    {
        /* only up to Ny/2 necessary, because wie use std::swap meaning we correct
         * two elements with 1 operation */
        for ( unsigned iy = 0; iy < Ny/2; ++iy )
        for ( unsigned ix = 0; ix < Nx; ++ix )
        {
            const unsigned index =
                ( ( iy+Ny/2 ) % Ny ) * Nx +
                ( ( ix+Nx/2 ) % Nx );
            std::swap( data[iy*Nx + ix], data[index] );
        }
    }


    /**
     * checks if the imaginary parts are all 0 for debugging purposes
     **/
    template< class T_COMPLEX >
    void checkIfReal
    (
        const T_COMPLEX * const & rData,
        const unsigned & rnElements
    )
    {
        float avgRe = 0;
        float avgIm = 0;

        for ( unsigned i = 0; i < rnElements; ++i )
        {
            avgRe += fabs( rData[i][0] );
            avgIm += fabs( rData[i][1] );
        }

        avgRe /= (float) rnElements;
        avgIm /= (float) rnElements;

        std::cout << std::scientific
                  << "Avg. Re = " << avgRe << "\n"
                  << "Avg. Im = " << avgIm << "\n";
        assert( avgIm <  1e-5 );
    }


    template< class T_PREC >
    float compareCpuWithGpuArray
    (
        const T_PREC * const & rpData,
        const T_PREC * const & rdpData,
        const unsigned & rnElements
    )
    {
        /* copy data from GPU in order to compare it */
        const unsigned nBytes = rnElements * sizeof(T_PREC);
        const T_PREC * const vec1 = rpData;
        T_PREC * const vec2 = (T_PREC*) malloc( nBytes );
        CUDA_ERROR( hipMemcpy( (void*) vec2, (void*) rdpData, nBytes, hipMemcpyDeviceToHost ) );

        float relErr = 0;

        //#pragma omp parallel for reduction( + : relErr )
        for ( unsigned i = 0; i < rnElements; ++i )
        {
            float max = fmax( fabs(vec1[i]), fabs(vec2[i]) );
            /* ignore 0/0 if both are equal and 0 */
            if ( max == 0 )
                max = 1;
            relErr += fabs( vec1[i] - vec2[i] ); // / max;
            //if ( i < 10 )
            //    std::cout << "    " << vec1[i] << " <-> " << vec2[i] << "\n";
        }

        free( vec2 );
        return relErr / rnElements;
    }


    /**
     *
     * In contrast to the normal hybrid input output this function takes
     * pointers to memory buffers instead of allocating them itself.
     * Furthermore it doesn't touch rIntensity and it returns F instead of f
     * in curData.
     * It also doesn't bother to calculate the error at each step.
     *
     * @param[in] rIntensity real measured intensity without phase
     * @param[in] rIntensityFirstGuess first guess for the phase of the
     *            intensity, e.g. a random phase
     * @param[in] gPrevious this isn't actually a guess for the object f, but
     *            an intermediary result for the HIO algorithm. For the first
     *            call it should be equal to g' = IFT[G == rIntensityFirstGuess]
     **/
    int cudaShrinkWrap
    (
        float * const & rIntensity,
        const std::vector<unsigned> & rSize,
        unsigned rnCycles,
        float rTargetError,
        float rHioBeta,
        float rIntensityCutOffAutoCorel,
        float rIntensityCutOff,
        float rSigma0,
        float rSigmaChange,
        unsigned rnHioCycles,
        unsigned rnCores
    )
    {
        if ( rSize.size() != 2 ) return 1;
        const unsigned & Ny = rSize[1];
        const unsigned & Nx = rSize[0];

        /* load libraries and functions which we need */
        using namespace imresh::algorithms;

        /* Evaluate input parameters and fill with default values if necessary */
        if ( rIntensity == NULL ) return 1;
        if ( rTargetError              <= 0 ) rTargetError              = 1e-5;
        if ( rnHioCycles               == 0 ) rnHioCycles               = 20;
        if ( rHioBeta                  <= 0 ) rHioBeta                  = 0.9;
        if ( rIntensityCutOffAutoCorel <= 0 ) rIntensityCutOffAutoCorel = 0.04;
        if ( rIntensityCutOff          <= 0 ) rIntensityCutOff          = 0.2;
        if ( rSigma0                   <= 0 ) rSigma0                   = 3.0;
        if ( rSigmaChange              <= 0 ) rSigmaChange              = 0.01;

        float sigma = rSigma0;

        /* calculate this (length of array) often needed value */
        unsigned nElements = 1;
        for ( unsigned i = 0; i < rSize.size(); ++i )
        {
            assert( rSize[i] > 0 );
            nElements *= rSize[i];
        }

        /* allocate needed memory so that HIO doesn't need to allocate and
         * deallocate on each call */
        hipfftComplex * dpCurData;
        hipfftComplex * dpgPrevious;
        float * dpIntensity;
        float * dpIsMasked;
        CUDA_ERROR( hipMalloc( (void**)&dpCurData  , sizeof(dpCurData  [0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpgPrevious, sizeof(dpgPrevious[0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpIntensity, sizeof(dpIntensity[0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpIsMasked , sizeof(dpIsMasked [0])*nElements ) );
        CUDA_ERROR( hipMemcpy( dpIntensity, rIntensity, sizeof(dpIntensity[0])*nElements, hipMemcpyHostToDevice ) );

        /* create fft plans G' to g' and g to G */
        hipfftHandle ftPlan;
        hipfftPlan2d( &ftPlan, Nx, Ny, HIPFFT_C2C );

        /* create first guess for mask from autocorrelation (fourier transform
         * of the intensity @see
         * https://en.wikipedia.org/wiki/Wiener%E2%80%93Khinchin_theorem */
        const unsigned nThreads = 512;
        const unsigned nBlocks  = ceil( (float) nElements / nThreads );
        cudaKernelCopyToRealPart<<< nBlocks, nThreads >>>( dpCurData, dpIntensity, nElements );

        hipfftExecC2C( ftPlan, dpCurData, dpCurData, HIPFFT_BACKWARD );
        cudaKernelComplexNormElementwise<<< nBlocks, nThreads >>>( dpIsMasked, dpCurData, nElements );
        //cudaFftShift( dpIsMasked, Nx,Ny );
        cudaGaussianBlur( dpIsMasked, Nx, Ny, sigma );

        /* apply threshold to make binary mask */
        const float maskedAbsMax = cudaVectorMax( dpIsMasked, nElements );
        const float maskedThreshold = rIntensityCutOffAutoCorel * maskedAbsMax;
        cudaKernelCutOff<<<nBlocks,nThreads>>>( dpIsMasked, nElements, maskedThreshold, 1.0f, 0.0f );

        /* copy original image into fftw_complex array @todo: add random phase */
        cudaKernelCopyToRealPart<<< nBlocks, nThreads >>>( dpCurData, dpIntensity, nElements );

        /* in the first step the last value for g is to be approximated
         * by g'. The last value for g, called g_k is needed, because
         * g_{k+1} = g_k - hioBeta * g' ! This is inside the loop
         * because the fft is needed */
        hipMemcpy( dpgPrevious, dpCurData, sizeof(dpCurData[0]) * nElements,
                    hipMemcpyDeviceToDevice );

        /* repeatedly call HIO algorithm and change mask */
        for ( unsigned iCycleShrinkWrap = 0; iCycleShrinkWrap < rnCycles; ++iCycleShrinkWrap )
        {
            /************************** Update Mask ***************************/
            std::cout << "Update Mask with sigma=" << sigma << "\n";

            /* blur |g'| (normally g' should be real!, so |.| not necessary) */
            cudaKernelComplexNormElementwise<<<nBlocks,nThreads>>>( dpIsMasked, dpCurData, nElements );
            cudaGaussianBlur( dpIsMasked, Nx, Ny, sigma );

            /* apply threshold to make binary mask */
            const float absMax = cudaVectorMax( dpIsMasked, nElements );
            const float threshold = rIntensityCutOff * absMax;
            cudaKernelCutOff<<<nBlocks,nThreads>>>( dpIsMasked, nElements, threshold, 1.0f, 0.0f );

            /* update the blurring sigma */
            sigma = fmax( 1.5f, ( 1.0f - rSigmaChange ) * sigma );

            for ( unsigned iHioCycle = 0; iHioCycle < rnHioCycles; ++iHioCycle )
            {
                /* apply domain constraints to g' to get g */
                cudaKernelApplyHioDomainConstraints<<<nBlocks,nThreads>>>
                    ( dpgPrevious, dpCurData, dpIsMasked, nElements, rHioBeta );

                /* Transform new guess g for f back into frequency space G' */
                hipfftExecC2C( ftPlan, dpgPrevious, dpCurData, HIPFFT_FORWARD );

                /* Replace absolute of G' with measured absolute |F| */
                cudaKernelApplyComplexModulus<<<nBlocks,nThreads>>>
                    ( dpCurData, dpCurData, dpIntensity, nElements );

                hipfftExecC2C( ftPlan, dpCurData, dpCurData, HIPFFT_BACKWARD );
            } // HIO loop

            /* check if we are done */
            const float currentError = calculateHioError( dpCurData /*g'*/, dpIsMasked, nElements );
            std::cout << "[Error " << currentError << "/" << rTargetError << "] "
                      << "[Cycle " << iCycleShrinkWrap << "/" << rnCycles-1 << "]"
                      << "\n";
            if ( rTargetError > 0 && currentError < rTargetError )
                break;
            if ( iCycleShrinkWrap >= rnCycles )
                break;
        } // shrink wrap loop
        cudaKernelCopyFromRealPart<<< nBlocks, nThreads >>>( dpIntensity, dpCurData, nElements );
        CUDA_ERROR( hipMemcpy( rIntensity, dpIntensity, sizeof(rIntensity[0])*nElements, hipMemcpyDeviceToHost ) );

        /* free buffers and plans */
        hipfftDestroy( ftPlan );
        CUDA_ERROR( hipFree( dpCurData   ) );
        CUDA_ERROR( hipFree( dpgPrevious ) );
        CUDA_ERROR( hipFree( dpIntensity ) );
        CUDA_ERROR( hipFree( dpIsMasked  ) );

        return 0;
    }

    /**
     * Same as cudaShrinkWrap but with support for async calls.
     */
    int shrinkWrap
    (
        float* const& rIntensity,
        const std::pair<unsigned,unsigned>& rSize,
        hipStream_t strm,
        unsigned rnCycles,
        float rTargetError,
        float rHioBeta,
        float rIntensityCutOffAutoCorel,
        float rIntensityCutOff,
        float rSigma0,
        float rSigmaChange,
        unsigned rnHioCycles,
        unsigned rnCores
    )
    {
        const unsigned& Ny = rSize.second;
        const unsigned& Nx = rSize.first;

        /* load libraries and functions which we need */
        using namespace imresh::algorithms;

        /* Evaluate input parameters and fill with default values if necessary */
        if ( rIntensity == NULL ) return 1;
        if ( rTargetError              <= 0 ) rTargetError              = 1e-5;
        if ( rnHioCycles               == 0 ) rnHioCycles               = 20;
        if ( rHioBeta                  <= 0 ) rHioBeta                  = 0.9;
        if ( rIntensityCutOffAutoCorel <= 0 ) rIntensityCutOffAutoCorel = 0.04;
        if ( rIntensityCutOff          <= 0 ) rIntensityCutOff          = 0.2;
        if ( rSigma0                   <= 0 ) rSigma0                   = 3.0;
        if ( rSigmaChange              <= 0 ) rSigmaChange              = 0.01;

        float sigma = rSigma0;

        /* calculate this (length of array) often needed value */
        assert( rSize.first > 0 && rSize.second > 0 );
        unsigned int nElements = 1 * rSize.first * rSize.second;

        /* allocate needed memory so that HIO doesn't need to allocate and
         * deallocate on each call */
        hipfftComplex * dpCurData;
        hipfftComplex * dpgPrevious;
        float * dpIntensity;
        float * dpIsMasked;
        CUDA_ERROR( hipMalloc( (void**)&dpCurData  , sizeof(dpCurData  [0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpgPrevious, sizeof(dpgPrevious[0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpIntensity, sizeof(dpIntensity[0])*nElements ) );
        CUDA_ERROR( hipMalloc( (void**)&dpIsMasked , sizeof(dpIsMasked [0])*nElements ) );
        CUDA_ERROR( hipMemcpyAsync( dpIntensity, rIntensity, sizeof(dpIntensity[0])*nElements, hipMemcpyHostToDevice, strm ) );

        /* create fft plans G' to g' and g to G */
        hipfftHandle ftPlan;
        hipfftPlan2d( &ftPlan, Nx, Ny, HIPFFT_C2C );

        /* create first guess for mask from autocorrelation (fourier transform
         * of the intensity @see
         * https://en.wikipedia.org/wiki/Wiener%E2%80%93Khinchin_theorem */
        const unsigned nThreads = 512;
        const unsigned nBlocks  = ceil( (float) nElements / nThreads );
        cudaKernelCopyToRealPart<<<nBlocks,nThreads,0,strm>>>( dpCurData, dpIntensity, nElements );

        hipfftExecC2C( ftPlan, dpCurData, dpCurData, HIPFFT_BACKWARD );
        cudaKernelComplexNormElementwise<<<nBlocks,nThreads,0,strm>>>( dpIsMasked, dpCurData, nElements );
        //cudaFftShift( dpIsMasked, Nx,Ny );
        cudaGaussianBlur( dpIsMasked, Nx, Ny, sigma );

        /* apply threshold to make binary mask */
        const float maskedAbsMax = cudaVectorMax( dpIsMasked, nElements );
        const float maskedThreshold = rIntensityCutOffAutoCorel * maskedAbsMax;
        cudaKernelCutOff<<<nBlocks,nThreads,0,strm>>>( dpIsMasked, nElements, maskedThreshold, 1.0f, 0.0f );

        /* copy original image into fftw_complex array @todo: add random phase */
        cudaKernelCopyToRealPart<<<nBlocks,nThreads,0,strm>>>( dpCurData, dpIntensity, nElements );

        /* in the first step the last value for g is to be approximated
         * by g'. The last value for g, called g_k is needed, because
         * g_{k+1} = g_k - hioBeta * g' ! This is inside the loop
         * because the fft is needed */
        hipMemcpyAsync( dpgPrevious, dpCurData, sizeof(dpCurData[0]) * nElements,
                    hipMemcpyDeviceToDevice, strm );

        /* repeatedly call HIO algorithm and change mask */
        for ( unsigned iCycleShrinkWrap = 0; iCycleShrinkWrap < rnCycles; ++iCycleShrinkWrap )
        {
            /************************** Update Mask ***************************/
#           ifdef IMRESH_DEBUG
                std::cout << "imresh::algorithms::cuda::shrinkWrap(): Update Mask with sigma=" << sigma << std::endl;
#           endif

            /* blur |g'| (normally g' should be real!, so |.| not necessary) */
            cudaKernelComplexNormElementwise<<<nBlocks,nThreads,0,strm>>>( dpIsMasked, dpCurData, nElements );
            cudaGaussianBlur( dpIsMasked, Nx, Ny, sigma );

            /* apply threshold to make binary mask */
            const float absMax = cudaVectorMax( dpIsMasked, nElements );
            const float threshold = rIntensityCutOff * absMax;
            cudaKernelCutOff<<<nBlocks,nThreads,0,strm>>>( dpIsMasked, nElements, threshold, 1.0f, 0.0f );

            /* update the blurring sigma */
            sigma = fmax( 1.5f, ( 1.0f - rSigmaChange ) * sigma );

            for ( unsigned iHioCycle = 0; iHioCycle < rnHioCycles; ++iHioCycle )
            {
                /* apply domain constraints to g' to get g */
                cudaKernelApplyHioDomainConstraints<<<nBlocks,nThreads,0,strm>>>
                    ( dpgPrevious, dpCurData, dpIsMasked, nElements, rHioBeta );

                /* Transform new guess g for f back into frequency space G' */
                hipfftExecC2C( ftPlan, dpgPrevious, dpCurData, HIPFFT_FORWARD );

                /* Replace absolute of G' with measured absolute |F| */
                cudaKernelApplyComplexModulus<<<nBlocks,nThreads,0,strm>>>
                    ( dpCurData, dpCurData, dpIntensity, nElements );

                hipfftExecC2C( ftPlan, dpCurData, dpCurData, HIPFFT_BACKWARD );
            } // HIO loop

            /* check if we are done */
            const float currentError = calculateHioError( dpCurData /*g'*/, dpIsMasked, nElements );
#           ifdef IMRESH_DEBUG
                std::cout << "imresh::algorithms::cuda::shrinkWrap(): [Error " <<
                    currentError << "/" << rTargetError << "] "
                    << "[Cycle " << iCycleShrinkWrap << "/" << rnCycles-1 << "]"
                    << std::endl;
#           endif
            if ( rTargetError > 0 && currentError < rTargetError )
                break;
            if ( iCycleShrinkWrap >= rnCycles )
                break;
        } // shrink wrap loop
        cudaKernelCopyFromRealPart<<<nBlocks,nThreads,0,strm>>>( dpIntensity, dpCurData, nElements );
        CUDA_ERROR( hipMemcpyAsync( rIntensity, dpIntensity, sizeof(rIntensity[0])*nElements, hipMemcpyDeviceToHost, strm ) );

        /* free buffers and plans */
        hipfftDestroy( ftPlan );
        CUDA_ERROR( hipFree( dpCurData   ) );
        CUDA_ERROR( hipFree( dpgPrevious ) );
        CUDA_ERROR( hipFree( dpIntensity ) );
        CUDA_ERROR( hipFree( dpIsMasked  ) );

        return 0;
    }
} // namespace cuda
} // namespace algorithms
} // namespace imresh
